#include "raylib.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include "renderer.hpp"
#include "interaction.hpp"
#include "timer.hpp"
#include "particle_simulation.cuh"




int main() {
    const int screenWidth = 800;
    const int screenHeight = 600;
    const int numParticles = 1000;
    const float influenceRadius = 150.0f; // Rayon d'influence de la souris
    const float targetRadius = 20.0f;     // Rayon de la cible



    // Initialiser la fen�tre
    InitGameWindow(screenWidth, screenHeight);

    // Initialiser le timer
    Timer timer(30.0f);

    // Sound
    InitAudioDevice();
    Music music = LoadMusicStream("hyper.mp3");
    PlayMusicStream(music);

    // Initialiser les particules sur le GPU
    Particle* deviceParticles = InitializeParticlesGPU(numParticles, screenWidth, screenHeight);

    // Initialisation du score sur le GPU
    int* deviceScore;
    int hostScore = 0;
    hipMalloc(&deviceScore, sizeof(int));
    hipMemcpy(deviceScore, &hostScore, sizeof(int), hipMemcpyHostToDevice);

    // Position de la cible (centr�e au d�but)
    float targetX = screenWidth / 2.0f;
    float targetY = screenHeight / 2.0f;

    // Vitesse initiale
    float speed = 1.0f;

    // D�tection de victoire
    bool victory = false;

    // Boucle principale
    while (!WindowShouldClose() && !victory) {
        float mouseX, mouseY;
        bool attract = false, repel = false;

        // Mettre � jour le timer
        timer.Update();
        

        // Maj lecture musique
        UpdateMusicStream(music);

        // G�rer les entr�es utilisateur (vitesse, position de la souris, etc.)
        ProcessUserInput(speed, mouseX, mouseY, attract, repel);

        // Mise � jour des particules avec CUDA
        int blockSize = 256;
        int numBlocks = (numParticles + blockSize - 1) / blockSize;
        UpdateParticles(deviceParticles, numParticles, mouseX, mouseY, targetX, targetY, targetRadius,
            attract, influenceRadius, deviceScore, speed);

        // Copier le score pour v�rifier la victoire
        hipMemcpy(&hostScore, deviceScore, sizeof(int), hipMemcpyDeviceToHost);

        // V�rifier la condition de victoire
        if (hostScore >= numParticles) {
            victory = true;
        }

        // Affichage
        BeginDrawing();
        ClearBackground(BLACK);

        // Dessiner la cible
        DrawCircle((int)targetX, (int)targetY, targetRadius, RED);

        // Dessiner les particules
        std::vector<Particle> hostParticles(numParticles);
        hipMemcpy(hostParticles.data(), deviceParticles, numParticles * sizeof(Particle), hipMemcpyDeviceToHost);
        for (const auto& particle : hostParticles) {
            if (particle.active) {
                DrawCircle((int)particle.x, (int)particle.y, 2.0f, { particle.r, particle.g, particle.b, particle.a });
            }
        }

        // Afficher le score et la vitesse
        DrawText(TextFormat("Score: %d", hostScore), 10, 10, 20, WHITE);
        DrawText(TextFormat("Speed: %.2f", speed), 10, 40, 20, GRAY);
        DrawText("Cible rouge: attirer les particules | Haut/Bas: changer vitesse", 10, 70, 20, GRAY);
        DrawText(timer.GetTimeLeft().c_str(), 10, 100, 20, WHITE);

        EndDrawing();
    }

    // Affichage de la victoire
    if (victory) {
        while (!WindowShouldClose()) {
            DrawVictoryScreen(screenWidth, screenHeight);
        }
    }

    // Lib�rer la m�moire GPU
    hipFree(deviceParticles);
    hipFree(deviceScore);

    // Unload music stream buffers from RAM
    UnloadMusicStream(music);   

    // Close audio device (music streaming is automatically stopped)
    CloseAudioDevice();       

    CloseWindow();
    return 0;
}
