#include "hip/hip_runtime.h"
#include "particle_simulation.cuh"

__global__ void UpdateParticlesWithMotion(Particle* particles, int numParticles, Obstacle* obstacles,
    int numObstacles, float mouseX, float mouseY, float targetX,
    float targetY, float targetRadius, bool attract,
    float influenceRadius, int* score, float speed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numParticles && particles[idx].active) {
        float nextX = particles[idx].x; // Position pr�vue en X
        float nextY = particles[idx].y; // Position pr�vue en Y

        // Influence de la souris (attraction ou r�pulsion)
        float dxMouse = mouseX - particles[idx].x;
        float dyMouse = mouseY - particles[idx].y;
        float mouseDistance = sqrtf(dxMouse * dxMouse + dyMouse * dyMouse);

        if (mouseDistance < influenceRadius && mouseDistance > 0.1f) {
            float factor = attract ? speed : -speed;
            nextX += factor * dxMouse / mouseDistance; // D�placement pr�vu en X
            nextY += factor * dyMouse / mouseDistance; // D�placement pr�vu en Y
        }
        else {
            // Mouvement normal
            nextX += particles[idx].dx * speed;
            nextY += particles[idx].dy * speed;
        }

        // V�rifier si la particule touche la cible
        float dxTarget = targetX - nextX;
        float dyTarget = targetY - nextY;
        float targetDistance = sqrtf(dxTarget * dxTarget + dyTarget * dyTarget);

        if (targetDistance < targetRadius) {
            particles[idx].active = false; // D�sactiver la particule
            atomicAdd(score, 1);          // Marquer un point
            return;                       // Arr�ter le traitement pour cette particule
        }

        // V�rification des collisions avec les obstacles
        bool collision = false;
        for (int i = 0; i < numObstacles; i++) {
            Obstacle obs = obstacles[i];
            if (nextX > obs.x && nextX < obs.x + obs.width &&
                nextY > obs.y && nextY < obs.y + obs.height) {
                collision = true;
                break;
            }
        }

        // Appliquer le mouvement uniquement s'il n'y a pas de collision
        if (!collision) {
            particles[idx].x = nextX;
            particles[idx].y = nextY;
        }
        else {
            // G�rer le rebond en cas de collision
            particles[idx].dx *= -1.0f;
            particles[idx].dy *= -1.0f;
        }

        // Gestion des bords de l'�cran
        if (particles[idx].x < BORDER_OFFSET || particles[idx].x > 800- BORDER_OFFSET) particles[idx].dx *= -1.0f;
        if (particles[idx].y < BORDER_OFFSET || particles[idx].y > 600- BORDER_OFFSET) particles[idx].dy *= -1.0f;
    }
}

// Fonction pour initialiser les particules sur le GPU
Particle* InitializeParticlesGPU(int numParticles, int screenWidth, int screenHeight,
    Obstacle* obstacles, int numObstacles) {
    std::vector<Particle> hostParticles(numParticles);

    for (int i = 0; i < numParticles; i++) {
        bool validPosition = false;

        while (!validPosition) {
            // G�n�rer une position al�atoire pour la particule
            float x = static_cast<float>(rand() % screenWidth);
            float y = static_cast<float>(rand() % screenHeight);

            // V�rifier si la position est valide (pas dans un obstacle)
            validPosition = true;
            for (int j = 0; j < numObstacles; j++) {
                Obstacle obs = obstacles[j];
                if (x > obs.x && x < obs.x + obs.width &&
                    y > obs.y && y < obs.y + obs.height) {
                    validPosition = false; // La position est invalide
                    break;
                }
            }

            // Si la position est valide, assigner les coordonn�es
            if (validPosition) {
                float angle = static_cast<float>(rand() % 360) * 3.14159f / 180.0f; // Angle al�atoire
                hostParticles[i] = {
                    x, y,                              // Position
                    cosf(angle) * 0.5f,               // Direction en X
                    sinf(angle) * 0.5f,               // Direction en Y
                    static_cast<unsigned char>(rand() % 256), // Couleur R
                    static_cast<unsigned char>(rand() % 256), // Couleur G
                    static_cast<unsigned char>(rand() % 256), // Couleur B
                    255,                              // Couleur A
                    true                              // Particule active
                };
            }
        }
    }

    // Copier les donn�es vers le GPU
    Particle* deviceParticles;
    hipMalloc(&deviceParticles, numParticles * sizeof(Particle));
    hipMemcpy(deviceParticles, hostParticles.data(), numParticles * sizeof(Particle), hipMemcpyHostToDevice);

    return deviceParticles;
}

// Fonction pour mettre � jour les particules
void UpdateParticles(Particle* deviceParticles, int numParticles, Obstacle* deviceObstacles, int numObstacles,
    float mouseX, float mouseY, float targetX, float targetY, float targetRadius, bool attract,
    float influenceRadius, int* deviceScore, float speed) {
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;

    UpdateParticlesWithMotion << <numBlocks, blockSize >> > (deviceParticles, numParticles, deviceObstacles,
        numObstacles, mouseX, mouseY, targetX, targetY,
        targetRadius, attract, influenceRadius, deviceScore, speed);
}
