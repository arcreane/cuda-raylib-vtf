#include "hip/hip_runtime.h"
#include "particle_simulation.cuh"

__global__ void UpdateParticlesWithMotion(Particle* particles, int numParticles, Obstacle* obstacles,
    int numObstacles, float mouseX, float mouseY, float targetX,
    float targetY, float targetRadius, bool attract,
    float influenceRadius, int* score, float speed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numParticles && particles[idx].active) {
        float nextX = particles[idx].x; // Position pr�vue en X
        float nextY = particles[idx].y; // Position pr�vue en Y

        // Influence de la souris (attraction ou r�pulsion)
        float dxMouse = mouseX - particles[idx].x;
        float dyMouse = mouseY - particles[idx].y;
        float mouseDistance = sqrtf(dxMouse * dxMouse + dyMouse * dyMouse);

        if (mouseDistance < influenceRadius && mouseDistance > 0.1f) {
            float factor = attract ? speed : -speed;
            nextX += factor * dxMouse / mouseDistance; // D�placement pr�vu en X
            nextY += factor * dyMouse / mouseDistance; // D�placement pr�vu en Y
        }
        else {
            // Mouvement normal
            nextX += particles[idx].dx * speed;
            nextY += particles[idx].dy * speed;
        }

        // V�rifier si la particule touche la cible
        float dxTarget = targetX - nextX;
        float dyTarget = targetY - nextY;
        float targetDistance = sqrtf(dxTarget * dxTarget + dyTarget * dyTarget);

        if (targetDistance < targetRadius) {
            particles[idx].active = false; // D�sactiver la particule
            atomicAdd(score, 1);          // Marquer un point
            return;                       // Arr�ter le traitement pour cette particule
        }

        // V�rification des collisions avec les obstacles
        bool collision = false;
        for (int i = 0; i < numObstacles; i++) {
            Obstacle obs = obstacles[i];
            if (nextX > obs.x && nextX < obs.x + obs.width &&
                nextY > obs.y && nextY < obs.y + obs.height) {
                collision = true;
                break;
            }
        }

        // Appliquer le mouvement uniquement s'il n'y a pas de collision
        if (!collision) {
            particles[idx].x = nextX;
            particles[idx].y = nextY;
        }
        else {
            // G�rer le rebond en cas de collision
            particles[idx].dx *= -1.0f;
            particles[idx].dy *= -1.0f;
        }

        // Gestion des bords de l'�cran
        if (particles[idx].x < 0 || particles[idx].x > 800) particles[idx].dx *= -1.0f;
        if (particles[idx].y < 0 || particles[idx].y > 600) particles[idx].dy *= -1.0f;
    }
}

// Fonction pour initialiser les particules sur le GPU
Particle* InitializeParticlesGPU(int numParticles, int screenWidth, int screenHeight) {
    std::vector<Particle> hostParticles(numParticles);

    // Initialisation des particules sur le CPU
    for (int i = 0; i < numParticles; i++) {
        float angle = (float)(rand() % 360) * DEG2RAD; // Direction al�atoire
        hostParticles[i] = {
            (float)(rand() % screenWidth),
            (float)(rand() % screenHeight),
            cosf(angle) * 0.5f, // Mouvement en X
            sinf(angle) * 0.5f, // Mouvement en Y
            (unsigned char)(rand() % 256),
            (unsigned char)(rand() % 256),
            (unsigned char)(rand() % 256),
            255,
            true
        };
    }

    // Copier les donn�es vers le GPU
    Particle* deviceParticles;
    hipMalloc(&deviceParticles, numParticles * sizeof(Particle));
    hipMemcpy(deviceParticles, hostParticles.data(), numParticles * sizeof(Particle), hipMemcpyHostToDevice);

    return deviceParticles;
}

// Fonction pour mettre � jour les particules
void UpdateParticles(Particle* deviceParticles, int numParticles, Obstacle* deviceObstacles, int numObstacles,
    float mouseX, float mouseY, float targetX, float targetY, float targetRadius, bool attract,
    float influenceRadius, int* deviceScore, float speed) {
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;

    UpdateParticlesWithMotion << <numBlocks, blockSize >> > (deviceParticles, numParticles, deviceObstacles,
        numObstacles, mouseX, mouseY, targetX, targetY,
        targetRadius, attract, influenceRadius, deviceScore, speed);
}
